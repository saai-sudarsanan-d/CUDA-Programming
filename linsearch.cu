#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__constant__ int key;

__global__ void linsearch(int *a,int *p){
    int t = threadIdx.x;
    if(a[t] == key){
        *p = t;
    }
}
int main(){
    int n,k,*darr,*p,pos=-1;
    printf("Enter the Number of Elements : ");
    scanf("%d",&n);
    int arr[n];
    printf("Enter the Elements : \n");
    for(int i=0;i<n;i++){
        scanf("%d",&arr[i]);
    }
    printf("\nEnter the element to search for : ");
    scanf("%d",&k);
    hipMalloc((void**)&darr,n*sizeof(int));
    hipMemcpy(darr,&arr,n*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc((void**)&p,sizeof(int));
    hipMemcpy(p,&pos,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(key),k, sizeof(int));
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    linsearch<<<1,n>>>(darr,p);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(&pos,p,sizeof(int),hipMemcpyDeviceToHost);
    printf("Element Found At : %d\n",pos);
    hipFree(p);
    hipFree(darr);
    return 0;
}