#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void prime(int *arr,int n){
    int t = threadIdx.x + blockIdx.x*blockDim.x;
    int flag = 1;
    for(int i=2;i<(int)t/2;i++){
        if(t%i == 0){
            flag = 0;
        }
    }
    if(t < 2){
        flag = 0;
    }
    if(t == 2){
        flag = 1;
    }
    if(flag == 1){
        arr[t] = 1;
    }
}
int main(){
    int n;
    int *da,arr[n];
    float ms;
    hipMalloc((void**)&da,sizeof(int)*n);
    hipMemcpy(da,&arr,sizeof(int)*n,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    prefix<<<5,n>>>(da);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(&arr,da,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("Finished in %f milliseconds\n",ms);
    printf("Prime Number Distribution from 0 to %d\n",n);
    for(int i=0;i<n;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
    return 0;
}
