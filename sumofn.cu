#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sumn(int *arr,int *res){
    int t = blockIdx.x*blockDim.x + threadIdx.x;
    atomicAdd(res,arr[t]);
}
int main(){
    int n = 100;
    int *da,*res,result,arr[n];
    for(int i=0;i<n;i++){
        arr[n] = i+1;
    }
    float ms;
    hipMalloc((void**)&da,sizeof(int)*n);
    hipMalloc((void**)&res,sizeof(int));
    hipMemcpy(da,&arr,sizeof(int)*n,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    sumn<<<2,50>>>(da,res);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(&result,res,sizeof(int),hipMemcpyDeviceToHost);
    printf("Finished in %f milliseconds\n",ms);
    printf("Sum is %d\n",result);
    printf("\n");
    return 0;
}