#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

__global__ void suffix(int *arr){
    int t = threadIdx.x;
    for(int i=0;i<2;i++){
        if(t + (int)pow(2,i) < 8){
            arr[t] = arr[t] + arr[t + (int)pow(2,i)];
        }
    }
}
int main(){
    int *da,arr[8] = {2,5,4,3,6,8,7,9};
    float ms;
    hipMalloc((void**)&da,sizeof(int)*8);
    hipMemcpy(da,&arr,sizeof(int)*8,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    prefix<<<1,8>>>(da);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(&arr,da,sizeof(int)*8,hipMemcpyDeviceToHost);
    printf("Finished in %f milliseconds\n",ms);
    for(int i=0;i<8;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
    return 0;
}