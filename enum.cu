#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void enumsort(int *arr){
    int t = threadIdx.x;
    int q;
    int c=0;
    for(int i=0;i<8;i++){
        if(t!=i && arr[t] > arr[i]){
            c++;
        }
    }
    q = arr[t];
    __syncthreads();
    arr[c] = q; 
}
int main(){
    float ms;
    int *da;
    int arr[8] = {2,3,5,4,6,8,7,9};
    hipMalloc((void**)&da,sizeof(int)*8);
    hipMemcpy(da,&arr,sizeof(int)*8,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    enumsort<<<1,8>>>(da);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    printf("Finished in %f ms\n",ms);
    printf("Final Array : ");
    hipMemcpy(&arr,da,sizeof(int)*8,hipMemcpyDeviceToHost);
    for(int i=0;i<8;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
}