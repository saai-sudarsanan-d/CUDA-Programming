#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void parrred(int *arr){
    int t = threadIdx.x;
    for(int i=0;i<8;i*=2){
        if(t%(2*i) == 0 && t+i < 8){
            arr[t] = arr[t] + arr[t+(2*i)];
        }
    }
    __syncthreads();
}
int main(){
    int *da,arr[8] = {1,2,3,4,5,6,7,8};
    float ms;
    hipMalloc((void**)&da,sizeof(int)*8);
    hipMemcpy(da,&arr,sizeof(int)*8,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    parrred<<<1,8>>>(da);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(&arr,da,sizeof(int)*8,hipMemcpyDeviceToHost);
    printf("Finished in %f milliseconds\n",ms);
    for(int i=0;i<8;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
    return 0;
}