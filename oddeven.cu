#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void oddeven(int *arr){
    int t = threadIdx.x;
    for(int i=0;i<4;i++){
        if(t%2 == 0 && t < 7){
            if(arr[t] > arr[t+1]){
                int temp = arr[t];
                arr[t] = arr[t+1];
                arr[t+1] = temp;
            }
        }
        __syncthreads();
        if(t%2 == 1 && t < 7){
            if(arr[t] > arr[t+1]){
                int temp = arr[t];
                arr[t] = arr[t+1];
                arr[t+1] = temp;
            }
        }
        __syncthreads();
    }
}
int main(){
    int *da,arr[8] = {2,5,4,3,6,8,7,9};
    float ms;
    hipMalloc((void**)&da,sizeof(int)*8);
    hipMemcpy(da,&arr,sizeof(int)*8,hipMemcpyHostToDevice);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    oddeven<<<1,8>>>(da);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(&arr,da,sizeof(int)*8,hipMemcpyDeviceToHost);
    printf("Finished in %f milliseconds\n",ms);
    for(int i=0;i<8;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
    return 0;
}